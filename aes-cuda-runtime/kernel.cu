#include "hip/hip_runtime.h"
﻿#include "functions.h"

/* ----- HELPERS ----- */

void printBytes(BYTE s[], int len) {
    for (int i = 0; i < len; i++)
        printf("%02x ", s[i]);
    printf("\n");
}

void printChars(BYTE s[], int len) {
    for (int i = 0; i < len; i++)
        printf("%c", s[i]);
    printf("\n");
}

void printToFile(BYTE s[], int len, FILE* fp) {
    for (int i = 0; i < len; i++) {
        fprintf(fp, "%c", s[i]);
    }
}

void generateIV(BYTE iv[], int ivSize) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 255);

    for (int i = 0; i < ivSize; i++) {
        iv[i] = static_cast<BYTE>(dis(gen));
    }
}

BYTE AES_Sbox_init[] =
{
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76,
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0,
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15,
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75,
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84,
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf,
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8,
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2,
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73,
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb,
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79,
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08,
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a,
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e,
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf,
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16
};

/* ----- AES FUNCTIONS ----- */

__device__ void AES_SubBytes(BYTE state[], BYTE sbox[]) {
    for (int i = 0; i < 16; i++)
        state[i] = sbox[state[i]];
}

__device__ void AES_AddRoundKey(BYTE state[], BYTE rkey[]) {
    for (int i = 0; i < 16; i++)
        state[i] ^= rkey[i];
}

__device__ void AES_ShiftRows(BYTE state[], BYTE shifttab[]) {
    BYTE temp[16];
    for (int i = 0; i < 16; i++)
        temp[i] = state[shifttab[i]];
    memcpy(state, temp, 16);
}

__device__ void AES_MixColumns(BYTE state[], BYTE AES_xtime[]) {
    for (int i = 0; i < 16; i += 4) {
        BYTE s0 = state[i + 0], s1 = state[i + 1];
        BYTE s2 = state[i + 2], s3 = state[i + 3];
        BYTE h = s0 ^ s1 ^ s2 ^ s3;
        state[i + 0] ^= h ^ AES_xtime[s0 ^ s1];
        state[i + 1] ^= h ^ AES_xtime[s1 ^ s2];
        state[i + 2] ^= h ^ AES_xtime[s2 ^ s3];
        state[i + 3] ^= h ^ AES_xtime[s3 ^ s0];
    }
}

__device__ void AES_MixColumns_Inv(BYTE state[], BYTE AES_xtime[]) {
    for (int i = 0; i < 16; i += 4) {
        BYTE s0 = state[i + 0], s1 = state[i + 1];
        BYTE s2 = state[i + 2], s3 = state[i + 3];
        BYTE h = s0 ^ s1 ^ s2 ^ s3;
        BYTE xh = AES_xtime[h];
        BYTE h1 = AES_xtime[AES_xtime[xh ^ s0 ^ s2]] ^ h;
        BYTE h2 = AES_xtime[AES_xtime[xh ^ s1 ^ s3]] ^ h;
        state[i + 0] ^= h1 ^ AES_xtime[s0 ^ s1];
        state[i + 1] ^= h2 ^ AES_xtime[s1 ^ s2];
        state[i + 2] ^= h1 ^ AES_xtime[s2 ^ s3];
        state[i + 3] ^= h2 ^ AES_xtime[s3 ^ s0];
    }
}

int AES_ExpandKey(BYTE key[], int keyLen, BYTE AES_Sbox[]) {
    int kl = keyLen, ks, Rcon = 1, i, j;
    BYTE* temp;

    switch (kl) {
    case 16: ks = 16 * (10 + 1); break;
    case 24: ks = 16 * (12 + 1); break;
    case 32: ks = 16 * (14 + 1); break;
    default:
        fprintf(stderr, "Error: Only 16, 24, or 32 key lengths are allowed.\n");
        return -1;
    }

    for (i = kl; i < ks; i += 4) {
        temp = &key[i - 4];
        if (i % kl == 0) {
            BYTE tmp = temp[0];
            temp[0] = AES_Sbox[temp[1]] ^ Rcon;
            temp[1] = AES_Sbox[temp[2]];
            temp[2] = AES_Sbox[temp[3]];
            temp[3] = AES_Sbox[tmp];
            if ((Rcon <<= 1) >= 256)
                Rcon ^= 0x11b;
        }
        else if ((kl > 24) && (i % kl == 16)) {
            for (j = 0; j < 4; j++)
                temp[j] = AES_Sbox[temp[j]];
        }
        for (j = 0; j < 4; j++)
            key[i + j] = key[i + j - kl] ^ temp[j];
    }
    return ks;
}

// AES_Encrypt & AES_Decrypt

__device__ void AES_Initialize_Encrypt(BYTE AES_Sbox_init[], BYTE AES_ShiftRowTab[], BYTE AES_xtime[]) {

    BYTE AES_Sbox[] =
    {
        0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76,
        0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0,
        0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15,
        0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75,
        0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84,
        0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf,
        0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8,
        0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2,
        0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73,
        0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb,
        0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79,
        0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08,
        0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a,
        0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e,
        0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf,
        0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16
    };

    int i;

    for (i = 0; i < 256; ++i) {
        AES_Sbox_init[i] = AES_Sbox[i];
    }

    for (i = 0; i < 16; ++i) {
        AES_ShiftRowTab[i] = (i % 4) * 4 + (i / 4);
    }

    for (i = 0; i < 128; i++) {
        AES_xtime[i] = i << 1;
        AES_xtime[128 + i] = (i << 1) ^ 0x1b;
    }
}

__device__ void AES_Initialize_Decrypt(BYTE AES_Sbox_Inv_init[], BYTE AES_ShiftRowTab_Inv[], BYTE AES_xtime[]) {

    BYTE AES_Sbox_Inv[256] =
    {
        0x52,0x09,0x6a,0xd5,0x30,0x36,0xa5,0x38,0xbf,0x40,0xa3,0x9e,0x81,0xf3,0xd7,0xfb,
        0x7c,0xe3,0x39,0x82,0x9b,0x2f,0xff,0x87,0x34,0x8e,0x43,0x44,0xc4,0xde,0xe9,0xcb,
        0x54,0x7b,0x94,0x32,0xa6,0xc2,0x23,0x3d,0xee,0x4c,0x95,0x0b,0x42,0xfa,0xc3,0x4e,
        0x08,0x2e,0xa1,0x66,0x28,0xd9,0x24,0xb2,0x76,0x5b,0xa2,0x49,0x6d,0x8b,0xd1,0x25,
        0x72,0xf8,0xf6,0x64,0x86,0x68,0x98,0x16,0xd4,0xa4,0x5c,0xcc,0x5d,0x65,0xb6,0x92,
        0x6c,0x70,0x48,0x50,0xfd,0xed,0xb9,0xda,0x5e,0x15,0x46,0x57,0xa7,0x8d,0x9d,0x84,
        0x90,0xd8,0xab,0x00,0x8c,0xbc,0xd3,0x0a,0xf7,0xe4,0x58,0x05,0xb8,0xb3,0x45,0x06,
        0xd0,0x2c,0x1e,0x8f,0xca,0x3f,0x0f,0x02,0xc1,0xaf,0xbd,0x03,0x01,0x13,0x8a,0x6b,
        0x3a,0x91,0x11,0x41,0x4f,0x67,0xdc,0xea,0x97,0xf2,0xcf,0xce,0xf0,0xb4,0xe6,0x73,
        0x96,0xac,0x74,0x22,0xe7,0xad,0x35,0x85,0xe2,0xf9,0x37,0xe8,0x1c,0x75,0xdf,0x6e,
        0x47,0xf1,0x1a,0x71,0x1d,0x29,0xc5,0x89,0x6f,0xb7,0x62,0x0e,0xaa,0x18,0xbe,0x1b,
        0xfc,0x56,0x3e,0x4b,0xc6,0xd2,0x79,0x20,0x9a,0xdb,0xc0,0xfe,0x78,0xcd,0x5a,0xf4,
        0x1f,0xdd,0xa8,0x33,0x88,0x07,0xc7,0x31,0xb1,0x12,0x10,0x59,0x27,0x80,0xec,0x5f,
        0x60,0x51,0x7f,0xa9,0x19,0xb5,0x4a,0x0d,0x2d,0xe5,0x7a,0x9f,0x93,0xc9,0x9c,0xef,
        0xa0,0xe0,0x3b,0x4d,0xae,0x2a,0xf5,0xb0,0xc8,0xeb,0xbb,0x3c,0x83,0x53,0x99,0x61,
        0x17,0x2b,0x04,0x7e,0xba,0x77,0xd6,0x26,0xe1,0x69,0x14,0x63,0x55,0x21,0x0c,0x7d
    };

    int i;

    for (i = 0; i < 256; ++i) {
        AES_Sbox_Inv_init[i] = AES_Sbox_Inv[i];
    }

    for (i = 0; i < 16; i++) {
        AES_ShiftRowTab_Inv[(i % 4) * 4 + (i / 4)] = i;
    }
        
    for (i = 0; i < 128; i++) {
        AES_xtime[i] = i << 1;
        AES_xtime[128 + i] = (i << 1) ^ 0x1b;
    }
}

__global__ void AES_Encrypt_ECB(AES_block aes_block_array[], BYTE key[], int keyLen, int block_number) {

    int gt_index = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ BYTE AES_ShiftRowTab[16];
    __shared__ BYTE AES_xtime[256];
    __shared__ BYTE AES_Sbox_init[256];

    AES_Initialize_Encrypt(AES_Sbox_init, AES_ShiftRowTab, AES_xtime);

    if (gt_index < block_number) {

        BYTE block[16];

        for (int i = 0; i < 16; i++) {
            block[i] = aes_block_array[gt_index].block[i];
        }

        __syncthreads();

        int l = keyLen, i;

        AES_AddRoundKey(block, &key[0]);
        for (i = 16; i < l - 16; i += 16) {
            AES_SubBytes(block, AES_Sbox_init);
            AES_ShiftRows(block, AES_ShiftRowTab);
            AES_MixColumns(block, AES_xtime);
            AES_AddRoundKey(block, &key[i]);
        }
        AES_SubBytes(block, AES_Sbox_init);
        AES_ShiftRows(block, AES_ShiftRowTab);
        AES_AddRoundKey(block, &key[i]);

        for (int i = 0; i < 16; i++) {
            aes_block_array[gt_index].block[i] = block[i];
        }

    }

}

__global__ void AES_Decrypt_ECB(AES_block aes_block_array[], BYTE key[], int keyLen, int block_number) {

    int gt_index = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ BYTE AES_ShiftRowTab_Inv[16];
    __shared__ BYTE AES_xtime[256];
    __shared__ BYTE AES_Sbox_Inv_init[256];

    AES_Initialize_Decrypt(AES_Sbox_Inv_init, AES_ShiftRowTab_Inv, AES_xtime);

    if (gt_index < block_number) {

        BYTE block[16];
        for (int i = 0; i < 16; i++) {
            block[i] = aes_block_array[gt_index].block[i];
        }

        __syncthreads();

        int l = keyLen, i;
        AES_AddRoundKey(block, &key[l - 16]);
        AES_ShiftRows(block, AES_ShiftRowTab_Inv);
        AES_SubBytes(block, AES_Sbox_Inv_init);
        for (i = l - 32; i >= 16; i -= 16) {
            AES_AddRoundKey(block, &key[i]);
            AES_MixColumns_Inv(block, AES_xtime);
            AES_ShiftRows(block, AES_ShiftRowTab_Inv);
            AES_SubBytes(block, AES_Sbox_Inv_init);
        }
        AES_AddRoundKey(block, &key[0]);

        for (int i = 0; i < 16; i++) {
            aes_block_array[gt_index].block[i] = block[i];
        }

    }
}

__global__ void AES_Encrypt_CBC(AES_block aes_block_array[], BYTE key[], int keyLen, BYTE iv[], int block_number) {
    int gt_index = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ BYTE AES_ShiftRowTab[16];
    __shared__ BYTE AES_xtime[256];
    __shared__ BYTE AES_Sbox_init[256];

    AES_Initialize_Encrypt(AES_Sbox_init, AES_ShiftRowTab, AES_xtime);

    if (gt_index < block_number) {
        BYTE block[16];
        BYTE prev_block[16];

        for (int i = 0; i < 16; i++) {
            block[i] = aes_block_array[gt_index].block[i];
            prev_block[i] = (gt_index == 0) ? iv[i] : aes_block_array[gt_index - 1].block[i];
        }

        __syncthreads();

        for (int i = 0; i < 16; i++) {
            block[i] ^= prev_block[i];
        }

        int l = keyLen, i;

        AES_AddRoundKey(block, &key[0]);
        for (i = 16; i < l - 16; i += 16) {
            AES_SubBytes(block, AES_Sbox_init);
            AES_ShiftRows(block, AES_ShiftRowTab);
            AES_MixColumns(block, AES_xtime);
            AES_AddRoundKey(block, &key[i]);
        }
        AES_SubBytes(block, AES_Sbox_init);
        AES_ShiftRows(block, AES_ShiftRowTab);
        AES_AddRoundKey(block, &key[i]);

        for (int i = 0; i < 16; i++) {
            aes_block_array[gt_index].block[i] = block[i];
        }
    }
}

__global__ void AES_Decrypt_CBC(AES_block aes_block_array[], BYTE key[], int keyLen, BYTE iv[], int block_number) {
    int gt_index = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ BYTE AES_ShiftRowTab_Inv[16];
    __shared__ BYTE AES_xtime[256];
    __shared__ BYTE AES_Sbox_Inv_init[256];

    AES_Initialize_Decrypt(AES_Sbox_Inv_init, AES_ShiftRowTab_Inv, AES_xtime);

    if (gt_index < block_number) {
        BYTE block[16];
        BYTE prev_block[16];

        for (int i = 0; i < 16; i++) {
            block[i] = aes_block_array[gt_index].block[i];
            prev_block[i] = (gt_index == 0) ? iv[i] : aes_block_array[gt_index - 1].block[i];
        }

        __syncthreads();

        int l = keyLen, i;
        AES_AddRoundKey(block, &key[l - 16]);
        AES_ShiftRows(block, AES_ShiftRowTab_Inv);
        AES_SubBytes(block, AES_Sbox_Inv_init);
        for (i = l - 32; i >= 16; i -= 16) {
            AES_AddRoundKey(block, &key[i]);
            AES_MixColumns_Inv(block, AES_xtime);
            AES_ShiftRows(block, AES_ShiftRowTab_Inv);
            AES_SubBytes(block, AES_Sbox_Inv_init);
        }
        AES_AddRoundKey(block, &key[0]);


        for (int i = 0; i < 16; i++) {
            block[i] ^= prev_block[i];
            aes_block_array[gt_index].block[i] = block[i];
        }
    }
}

void readBlocksFromFile(char* inputFile, AES_block*& aes_block_array, int& block_number, int& incomplete_block_number) {

    std::ifstream ifs;
    ifs.open(inputFile, std::ios::binary);

    if (!ifs) {
        std::cerr << "Cannot open the input file" << std::endl;
        exit(1);
    }

    ifs.seekg(0, std::ios::end);
    int fileLength = ifs.tellg();
    ifs.seekg(0, std::ios::beg);

    block_number = fileLength / 16;
    incomplete_block_number = fileLength % 16;

    if (incomplete_block_number != 0)
        aes_block_array = new AES_block[block_number + 1];
    else
        aes_block_array = new AES_block[block_number];
    char temp[16];

    // read blocks
    for (int i = 0; i < block_number; i++) {
        ifs.read(temp, 16);
        for (int j = 0; j < 16; j++) {
            aes_block_array[i].block[j] = (unsigned char)temp[j];
        }
    }

    // read incomplete blocks
    if (incomplete_block_number != 0) {
        ifs.read(temp, incomplete_block_number);
        for (int j = 0; j < 16; j++) {
            aes_block_array[block_number].block[j] = (unsigned char)temp[j];
        }
        for (int j = 1; j <= 16 - incomplete_block_number; j++)
            aes_block_array[block_number].block[16 - j] = 0x00;
        block_number++;
    }

    ifs.close();
}

void writeBlocksToFile(char* inputFile, AES_block* aes_block_array, int block_number, int incomplete_block_number) {

    FILE* file;
    file = fopen(inputFile, "wb");

    for (int i = 0; i < block_number ; i++) {
        printToFile(aes_block_array[i].block, 16, file);
    }

    fclose(file);

}

void cudaEncrypt(AES_block*& aes_block_array, BYTE key[], int expandKeyLen, BYTE iv[], int block_number) {

    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int nnm = prop.multiProcessorCount;
    AES_block* cuda_aes_block_array;
    BYTE* cuda_key;
    int thrdperblock = block_number / nnm;

    // cuda threads
    if (block_number % nnm > 0)
        thrdperblock++;
    if (thrdperblock > 1024) {
        thrdperblock = 1024;
        nnm = block_number / 1024;
        if (block_number % 1024 > 0) {
            nnm++;
        }
    }
    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(nnm);

    hipMalloc(&cuda_aes_block_array, block_number * sizeof(class AES_block));
    hipMalloc(&cuda_key, 16 * 15 * sizeof(BYTE));
    hipMemcpy(cuda_aes_block_array, aes_block_array, block_number * sizeof(class AES_block), hipMemcpyHostToDevice);
    hipMemcpy(cuda_key, key, 16 * 15 * sizeof(BYTE), hipMemcpyHostToDevice);

    AES_Encrypt_ECB << < BlockperGrid, ThreadperBlock >> > (cuda_aes_block_array, cuda_key, expandKeyLen, block_number);
    //AES_Encrypt_CBC << <BlockperGrid, ThreadperBlock >> > (cuda_aes_block_array, cuda_key, expandKeyLen, iv, block_number);
    hipMemcpy(aes_block_array, cuda_aes_block_array, block_number * sizeof(class AES_block), hipMemcpyDeviceToHost);

    hipFree(cuda_aes_block_array);
    hipFree(cuda_key);
}

void cudaDecrypt(AES_block*& aes_block_array, BYTE key[], int expandKeyLen, BYTE iv[], int block_number) {
    
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int nnm = prop.multiProcessorCount;
    AES_block* cuda_aes_block_array;
    BYTE* cuda_key;
    int thrdperblock = block_number / nnm;

    // cuda threads
    if (block_number % nnm > 0)
        thrdperblock++;
    if (thrdperblock > 1024) {
        thrdperblock = 1024;
        nnm = block_number / 1024;
        if (block_number % 1024 > 0) {
            nnm++;
        }
    }
    dim3 ThreadperBlock(thrdperblock);
    dim3 BlockperGrid(nnm);

    hipMalloc(&cuda_aes_block_array, block_number * sizeof(class AES_block));
    hipMalloc(&cuda_key, 16 * 15 * sizeof(BYTE));
    hipMemcpy(cuda_aes_block_array, aes_block_array, block_number * sizeof(class AES_block), hipMemcpyHostToDevice);
    hipMemcpy(cuda_key, key, 16 * 15 * sizeof(BYTE), hipMemcpyHostToDevice);

    AES_Decrypt_ECB << < BlockperGrid, ThreadperBlock >> > (cuda_aes_block_array, cuda_key, expandKeyLen, block_number);
    //AES_Decrypt_CBC << <BlockperGrid, ThreadperBlock >> > (cuda_aes_block_array, cuda_key, expandKeyLen, iv, block_number);
    hipMemcpy(aes_block_array, cuda_aes_block_array, block_number * sizeof(class AES_block), hipMemcpyDeviceToHost);

    hipFree(cuda_aes_block_array);
    hipFree(cuda_key);
}

void getKey(char* keyLine, BYTE key[16 * (14 + 1)], int& expandKeyLen) {
   
    /* ----- KEY ----- */

    int keyLen = 0;
    for (int i = 0; keyLine[i] != '\0'; ++i) {
        key[keyLen++] = keyLine[i];
    }
    expandKeyLen = AES_ExpandKey(key, keyLen, AES_Sbox_init);

}

AES_block * AES_Encrypt(char* keyLine, AES_block* aes_block_array, int block_number, BYTE iv[], int incomplete_block_number) {

    /* ----- ENCRYPTION ----- */

    BYTE key[16 * (14 + 1)];
    int expandKeyLen = 0;

    getKey(keyLine, key, expandKeyLen);

    generateIV(iv, sizeof(iv));

    //AES_Encrypt_ECB(aes_block_array, key, expandKeyLen, block_number);
    //AES_Encrypt_CBC(aes_block_array, key, expandKeyLen, iv, block_number);
    cudaEncrypt(aes_block_array, key, expandKeyLen, iv, block_number);

    return aes_block_array;
}

AES_block * AES_Decrypt(char* keyLine, AES_block* aes_block_array, int block_number, BYTE iv[], int incomplete_block_number) {

    /* ----- DECRYPTION ----- */

    BYTE key[16 * (14 + 1)];
    int expandKeyLen = 0;

    getKey(keyLine, key, expandKeyLen);

    //AES_Decrypt_ECB(aes_block_array, key, expandKeyLen, block_number);
    //AES_Decrypt_CBC(aes_block_array, key, expandKeyLen, iv, block_number);
    cudaDecrypt(aes_block_array, key, expandKeyLen, iv, block_number);

    return aes_block_array;
}